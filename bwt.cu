#include "hip/hip_runtime.h"
/*
 * bwt.cu
 *
 *  Created on: May 2, 2014
 *      Author: jiang
 */
#include <zlib.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <inttypes.h>
#include <thrust/version.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include "kseq.h"
#include "bwt.h"
// STEP 1: declare the type of file handler and the read() function
KSEQ_INIT(gzFile, gzread)
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
unsigned char nst_nt4_table[256] = {
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 5 /*'-'*/, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4
};
int fasta_parser(char *indexFile,Sequence &sequence)
{
	gzFile fp;
	kseq_t *seq;
	int l;
	fp = gzopen(indexFile, "r"); // STEP 2: open the file handler
	seq = kseq_init(fp); // STEP 3: initialize seq
	while ((l = kseq_read(seq)) >= 0) { // STEP 4: read sequence
		//printf("name: %s\n", seq->name.s);
		//if (seq->comment.l) printf("comment: %s\n", seq->comment.s);
		//printf("seq: %s\n", seq->seq.s);
		sequence.length = seq->seq.l;
		sequence.seq = (uint8_t*) calloc(sequence.length, 1);
		for (int i = 0; i != seq->seq.l; i++)
			sequence.seq[i] = nst_nt4_table[(int) seq->seq.s[i]];
		//if (seq->qual.l) printf("qual: %s\n", seq->qual.s);
	}
	printf("return value: %d\n", l);
	kseq_destroy(seq); // STEP 5: destroy seq
	gzclose(fp); // STEP 6: close the file handler
	return 0;
}



__global__ void count_suffixs(uint32_t *d_A ,const uint8_t *d_sequence,uint64_t l, int prefix_len,uint64_t seq_len,uint32_t prefix)
{
	uint32_t idx = threadIdx.x + blockDim.x *blockIdx.x;
	uint64_t start_pos = idx * l, end_pos = (idx+1) * l; //end_pos not included
	uint32_t count = 0;
	uint32_t value  =0;
	uint32_t debug_count= 0;
	//extern __shared__ uint32_t s_A[];
	if(idx == BLOCKS_NUMBER*THREADS_NUMBER-1) {// last thread
		end_pos = seq_len;
		//printf("thread  %d  is the last one!\n",idx);
	}
	//Each thread scans through start_pos to end_pos
	if(idx != BLOCKS_NUMBER*THREADS_NUMBER-1)
	{
		for(uint64_t i = 0 ; i< l;  i++){
			value = 0;
			for(int k = 0; k < prefix_len; k++){
					value <<= 2;
					value += (uint32_t)d_sequence[start_pos+k+i];
			}
			debug_count ++;
			if(value == prefix) count++;
		}
	}
	else{ //last thread
		for(uint64_t i = 0 ; i< l;  i++){
			if(start_pos + i <= end_pos - prefix_len) //all threads except for last part of last thread must satisfy
			{
				value = 0;
				for(int k = 0; k < prefix_len; k++){
						value <<= 2;
						value += (uint32_t)d_sequence[start_pos+k+i];
				}
				debug_count ++;
				if(value == prefix) count++;
			}
			else if ((start_pos + i > end_pos - prefix_len)  && (start_pos + i < end_pos))
			{ //The last part of last thread is reading end of the text
				int left = seq_len - (start_pos + i);
				value = 0;
				for(int k = 0; k < left; k++)
				{
					value <<= 2;
					value += (uint32_t)d_sequence[start_pos+i+k];
				}
				for(int k = 0; k < prefix_len -left; k++)
				{
					value <<= 2;
					value += (uint32_t)d_sequence[k];
				}
				debug_count ++;
				if(value == prefix) count++;
			}
			else break;
		}
	}
	d_A[idx] = count;
	//printf("thread  %d  run %d times\n",idx,debug_count);
   //printf("prefix is %d thread  %d  count %llu end_pos %d\n",prefix,threadIdx.x, count,end_pos);
}

__global__ void get_suffix_block(const uint32_t *d_A, const uint8_t *d_sequence,uint64_t l, int prefix_len,uint64_t seq_len,uint32_t prefix,uint32_t *d_B){
	uint32_t idx  = threadIdx.x + blockDim.x *blockIdx.x;
	//printf("thread  %d  value  %d\n",threadIdx.x,d_A[idx]);
	uint64_t start_pos = idx * l, end_pos = (idx+1) * l; //end_pos not included
	uint32_t b = 0;
	uint32_t value  =0;
	uint8_t tail_seq[10];
	uint32_t debug_count= 0;
	if(idx == BLOCKS_NUMBER*THREADS_NUMBER-1) {// last thread
			end_pos = seq_len;
	}
	//Each thread scans through start_pos to end_pos
		if(idx != BLOCKS_NUMBER*THREADS_NUMBER-1)
		{
			for(uint64_t i = 0 ; i< l;  i++){
				value = 0;
				for(int k = 0; k < prefix_len; k++){
						value <<= 2;
						value += (uint32_t)d_sequence[start_pos+k+i];
				}
				debug_count ++;
				if(value == prefix){
					d_B[d_A[idx] + b]= start_pos+i ;
					b++;

				}
			}
		}
		else{ //last thread
			for(uint64_t i = 0 ; i< l;  i++){
				if(start_pos + i <= end_pos - prefix_len) //all threads except for last part of last thread must satisfy
				{
					value = 0;
					for(int k = 0; k < prefix_len; k++){
							value <<= 2;
							value += (uint32_t)d_sequence[start_pos+k+i];
					}
					debug_count ++;
					if(value == prefix) {
						d_B[d_A[idx] + b]= start_pos+i ;
						b++;

					}
				}
				else if ((start_pos + i > end_pos - prefix_len)  && (start_pos + i < end_pos))
				{ //The last part of last thread is reading end of the text
					int left = seq_len - (start_pos + i);
					value = 0;
					for(int k = 0; k < left; k++)
					{
						value <<= 2;
						value += (uint32_t)d_sequence[start_pos+i+k];
					}
					for(int k = 0; k < prefix_len -left; k++)
					{
						value <<= 2;
						value += (uint32_t)d_sequence[k];
					}
					debug_count ++;
					if(value == prefix) {
						d_B[d_A[idx] + b]= start_pos+i ;
						b++;

					}
				}
				else break;
			}
		}



}

__global__ void map_pos2key(const uint8_t  *d_sequence,uint64_t seq_length,int prefix_len, int v_len, const uint32_t *d_B,uint32_t B_len, uint32_t *d_B_key)
{
	uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < B_len){
		uint64_t position = d_B[idx];
		uint32_t value = 0;
		if(position <= (uint64_t)(seq_length - (v_len+1))){
			for(int i = prefix_len; i < v_len; i++)
			{
				value <<= 2;
				value += (uint32_t)d_sequence[position + i];
			}

		}
		else
		{
			int left = seq_length - position;
			value = 0;
			for(int k = 0; k < left; k++)
			{
				value <<= 2;
				value += (uint32_t)d_sequence[position+k];
			}
			for(int k = 0; k < prefix_len -left; k++)
			{
				value <<= 2;
				value += (uint32_t)d_sequence[k];
			}

		}
		d_B_key[idx] = value;
	}

}
/*
__global__ void checking(uint32_t *d_B_key, uint32_t B_len)
{


}
*/
void suffix_blocking(uint32_t prefix, const uint8_t  *d_sequence, uint8_t *h_sequence,uint64_t seq_length,int prefix_len, FILE *bwt_file)
{

	 uint32_t *d_A = NULL; //counting array
	 uint32_t *h_A = NULL;
	 uint32_t *h_B = NULL;
	 uint32_t *d_B = NULL;
	 uint32_t *h_B_key = NULL;
	 uint32_t *d_B_key = NULL;

	 fprintf(stderr,"block %llu processing\n",prefix);
	 //printf("version %d %d",THRUST_MAJOR_VERSION, THRUST_MINOR_VERSION);
	 uint64_t l = seq_length/(THREADS_NUMBER*BLOCKS_NUMBER) +1;

	 //printf("l is %d\n",l);
	 //exit(1);

	 h_A = (uint32_t*) malloc(sizeof(uint32_t) * (THREADS_NUMBER*BLOCKS_NUMBER));
	 if(h_A == NULL){
		 fprintf(stderr,"Out of host memory!\n");

	 }

	 CUDA_CHECK_RETURN(hipMalloc((void**) &d_A, sizeof(uint32_t) * (THREADS_NUMBER*BLOCKS_NUMBER)));
	 count_suffixs<<<BLOCKS_NUMBER, THREADS_NUMBER>>>(d_A, d_sequence,l,prefix_len,seq_length, prefix);
	 CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	 //exit(1);
	 hipMemcpy(h_A , d_A, sizeof(uint32_t) * (THREADS_NUMBER*BLOCKS_NUMBER), hipMemcpyDeviceToHost);
	 uint32_t last = h_A[THREADS_NUMBER*BLOCKS_NUMBER-1];

	 //fprintf(stderr,"count suffix finish\n");
	 //exclusive prefix sum
	 thrust::exclusive_scan(h_A,h_A+THREADS_NUMBER*BLOCKS_NUMBER,h_A);

	 //fprintf(stderr,"scan finish\n");
	 /*
	 for(int i=0; i < THREADS_NUMBER; i++)
	 {
		 fprintf(stdout,"%d ",h_A[i]);
	 }
	 */



	 //calculate number of elments in B
	 uint32_t B_len = h_A[THREADS_NUMBER*BLOCKS_NUMBER-1] + last;
	 h_B = (uint32_t*) malloc(sizeof(uint32_t) * B_len);
	 if(h_B == NULL){
			 fprintf(stderr,"Out of host memory!\n");
			 exit(1);
	 }
	 CUDA_CHECK_RETURN(hipMalloc((void**) &d_B, sizeof(uint32_t) * B_len));
	 hipMemcpy(d_A , h_A, sizeof(uint32_t) * (THREADS_NUMBER*BLOCKS_NUMBER), hipMemcpyHostToDevice);
	 //fprintf(stdout,"\nB_len%d!\n",B_len);
	 get_suffix_block<<<BLOCKS_NUMBER, THREADS_NUMBER>>>(d_A, d_sequence,l, prefix_len, seq_length,prefix,d_B);
	 CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	 hipMemcpy(h_B , d_B, sizeof(uint32_t) * B_len, hipMemcpyDeviceToHost);
	 /*
	 fprintf(stdout,"\nh_B:");
	 for(int i=0; i < B_len; i++)
	 {
	 	fprintf(stdout,"%d ",h_B[i]);
	 }
	 */
	 CUDA_CHECK_RETURN(hipFree(d_A));
	 //fprintf(stderr,"cal member finish\n");

	 //Map position to a key
	 int thread_num =1024;
	 int block_num = B_len/thread_num +1 ;
	 h_B_key = (uint32_t*) malloc(sizeof(uint32_t) * B_len);
	 if(h_B_key == NULL){
			 fprintf(stderr,"Out of host memory!\n");
			 exit(1);
	 }
	 CUDA_CHECK_RETURN(hipMalloc((void**) &d_B_key, sizeof(uint32_t) * B_len));
	 int v_len = 16 + prefix_len; //assume we can use first v_len characters to distinguish suffixes
	 map_pos2key<<<block_num, thread_num>>>(d_sequence,seq_length,prefix_len, v_len,d_B, B_len,d_B_key);
	 hipMemcpy(h_B_key , d_B_key, sizeof(uint32_t) * B_len, hipMemcpyDeviceToHost);

	 /*
	 fprintf(stdout,"\n");
	 for(int i=0; i < B_len; i++)
	 {
	 	fprintf(stdout,"%llu ",h_B_key[i]);
	 }
	 */
	 //Radix sort
	 thrust::sort_by_key(h_B_key,h_B_key + B_len, h_B);
	 //fprintf(stderr,"radix sort finish\n");
	 /*
	 fprintf(stdout,"\nSorted");
	 for(int i=0; i < B_len; i++)
	 {
		 	fprintf(stdout,"(%llu,%llu) ",h_B[i],h_B_key[i]);
	 }
	 */

	 //Construct the BWT
	 //FIXME  Now h_B is almost the suffix array, possibly with some minor errors, since we need a refinement
	 uint8_t bwt_value;
	 for(uint32_t i =0; i<B_len; i++)
	 {
		 if(h_B[i]==0) bwt_value = h_sequence[seq_length -1];
		 else bwt_value = h_sequence[h_B[i]-1];
		 fseek(bwt_file, 0, SEEK_END);
		 fprintf(bwt_file, "%d",bwt_value);
	 }

	 //Parallel checking algorithm

}

int bwt(char *indexFile, uint32_t prefix_len){
	uint8_t *d_sequence =NULL;
	int forward_only = 1;
	uint64_t buf_length;
	Sequence sequence;
	fasta_parser(indexFile,sequence);
	//fprintf(stdout,"seqlength %d ",  sequence.length);
	/*
	fprintf(stdout,"seq:");
	for (int i = 0; i != sequence.length; i++)
	{
		fprintf(stdout,"%d",sequence.seq[i]);
	}
	fprintf(stdout,"\n");
	*/
	FILE *bwt_file;
	char *bwt_fn;
	bwt_fn = (char*) calloc(strlen(indexFile) + 10, 1);
	strcpy(bwt_fn, indexFile);
	strcat(bwt_fn, ".bwt");

	bwt_file = fopen(bwt_fn, "wb");


	CUDA_CHECK_RETURN(hipMalloc((void**) &d_sequence, sizeof(uint8_t) * sequence.length));
		CUDA_CHECK_RETURN(
			hipMemcpy(d_sequence, sequence.seq, sizeof(uint8_t) * sequence.length, hipMemcpyHostToDevice)); //copy text to global memory of GPU
	//There will be 4^prefix_len number of blocks
	uint32_t number_blocks = pow(4.0,(int)prefix_len);
	for(uint32_t p = 0; p < number_blocks; p++){
		  suffix_blocking(p,d_sequence,sequence.seq,sequence.length,prefix_len,bwt_file);
		  //exit(1);
	}
	return 0;
}
